#include "hip/hip_runtime.h"
#include "saxpy.c"
#include <stdio.h>
#include <assert.h>
#include <stdlib.h>
#include <sys/time.h>

__global__
void cuda_saxpy(int num_threads, int n, float a, float *d_x, float *d_y)
{
  int i = threadIdx.x;

  // This loop performs 3 floating point ops per iteration.
  while(i<n){
    d_y[i] = a*d_x[i] + d_y[i];
    i = i+num_threads;
  }
}

int main(int argc, char *argv[]){
  if ( argc != 3 ) /* argc should be 3 for correct execution */
    {
      /* We print argv[0] assuming it is the program name */
      printf( "usage: %s (int) <elements_in_vector> (int) <num_threads>\n", argv[0] );
      return 1;
    }

  // Var for error handling
  hipError_t err = hipSuccess;

  int n = atoi(argv[1]);
  int num_threads = atoi(argv[2]);
  float a = 2.0;

  // Size for memory transfers
  int size = sizeof(float)*n;

  // Seed rand
  srand (time(NULL));

  // Allocate arrays
  float *x = (float *)malloc(sizeof(float)*n);
  float *y = (float *)malloc(sizeof(float)*n);

  // Generate Random Arrays
  populateRandomFloatArray(n, x);
  populateRandomFloatArray(n, y);

  // Start the timer
  struct timeval tim;
  gettimeofday(&tim, NULL);
  double t1=tim.tv_sec+(tim.tv_usec/1000000.0);

  // Default to the first GPU
  err = hipSetDevice(0);

  if (err != hipSuccess){
    fprintf(stderr, "Failed to default to CUDA device 0! (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Allocate device memory
  float *d_x;
  float *d_y;
  err = hipMalloc((void **) &d_x, sizeof(float)*n);
  //  printf("DEBUG: hipMalloc d_x size = %d\n", sizeof(float)*n);
  
  if (err != hipSuccess){
    fprintf(stderr, "Failed to allocate device vector d_x (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  err = hipMalloc((void **) &d_y, sizeof(float)*n);

  if (err != hipSuccess){
    fprintf(stderr, "Failed to allocate device vector d_y (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Copy data into d_x
  err = hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
  if (err != hipSuccess){
    fprintf(stderr, "Failed to mem copy data into d_x (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Copy data into d_y
  err = hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);
  if (err != hipSuccess){
    fprintf(stderr, "Failed to mem copy data into d_y (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Perform CUDA SAXPY
  cuda_saxpy<<<1,num_threads>>>(num_threads, n, a, d_x, d_y);
  hipDeviceSynchronize();
  
  // Copy result back
  err = hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);
  if (err != hipSuccess){
    fprintf(stderr, "Failed to memcpy result back from device. (error code %s)!\n", hipGetErrorString(err));
    fprintf(stderr, "The memcpy size was: %d\n", size);
    exit(EXIT_FAILURE);
  }

  // Print timing information
  gettimeofday(&tim, NULL);
  double t2=tim.tv_sec+(tim.tv_usec/1000000.0);
  printf("%.6lf\t", (((2*n)/(t2-t1))/1000000)); // 1000000000 = 10^9, 1000000 = 10^6
  //printf("%d\t%d\t%.6lf\t", num_threads, n, t2-t1);

  // cpu free
  free(x);
  free(y);

  // cuda free
  err = hipFree(d_x);
  if (err != hipSuccess){
    fprintf(stderr, "Failed to free device memory d_x (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  err = hipFree(d_y);
  if (err != hipSuccess){
    fprintf(stderr, "Failed to free device memory d_y (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}
